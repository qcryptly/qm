#include "hip/hip_runtime.h"
#include "vector.h"
#include <iostream>
#include <math.h>

// Add vector
__global__
void _cuda_add(int n, float *x, float *y) {
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
}

namespace Vectors {
  // Accepts
  void add(float *&x, float *&y, void(*callable)(float *&, float *&, int), int N) {
    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    callable(x, y, N);

    // Run kernel on 1M elements on the GPU
    _cuda_add<<<1, 256>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

  }

  void free(float *&x, float *&y) {
    hipFree(x);
    hipFree(y);
  }
}
