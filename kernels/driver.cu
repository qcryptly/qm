#include "driver.h"

namespace KDriver {
    Device::Device() {
      hipGetDevice(&device_);
      hipGetDeviceProperties(&properties_, device_);
    }

    void Device::setKernel(void(*kernel)()) {
      kernel_ = kernel;
    }
}
