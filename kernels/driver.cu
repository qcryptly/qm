#include "hip/hip_runtime.h"
#include "driver.h"

__global__ void _default_kernel(float4 *pos, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // calculate uv coordinates
    float u = x / (float) width;
    float v = y / (float) height;
    u = u*2.0f - 1.0f;
    v = v*2.0f - 1.0f;

    // calculate simple sine wave pattern
    float freq = 4.0f;
    float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

    // write output vertex
    pos[y*width+x] = make_float4(u, w, v, 1.0f);
}

namespace KDriver {
    void defaultKernel (hipGraphicsResource **vbo_res_, dim3 grid, dim3 block, float time) {
      float4 *dptr;
      hipGraphicsMapResources(1, vbo_res_, 0);
      size_t num_bytes;
      hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
        *vbo_res_);
      _default_kernel<<<grid, block>>>(dptr, 256, 256, time);
      hipGraphicsUnmapResources(1, vbo_res_, 0);
    }

    Device::Device() {
      hipGetDevice(&device_);
      hipGetDeviceProperties(&properties_, device_);
      dim3 block(8, 8, 1);
      dim3 grid(256 / block.x, 256 / block.y, 1);
    }

    void Device::setDefaultKernel() {
      kernel_ = defaultKernel;
    }

    void Device::makeGLBuffer(unsigned& buffer) {
      hipGraphicsGLRegisterBuffer(&vbo_res_, buffer, cudaGraphicsMapFlagsWriteDiscard);
    }

    void Device::setGLDevice() {
      cudaGLSetGLDevice(device_);
    }

    void Device::run(float time) {
      kernel_(&vbo_res_, grid_, block_, time);
    }
}
