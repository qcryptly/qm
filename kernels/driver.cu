#include "hip/hip_runtime.h"
#include "driver.h"
#include <iostream>

__global__ void _default_kernel(float4 *pos, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // calculate uv coordinates
    float u = x / (float) width;
    float v = y / (float) height;
    u = u*2.0f - 1.0f;
    v = v*2.0f - 1.0f;

    // calculate simple sine wave pattern
    float freq = 4.0f;
    float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

    // write output vertex
    pos[y*width+x] = make_float4(0.0,0.0,0.0,0.0);//u, w, v, 0.0f);
}

void checkCudaErrors(int cudaStatus) {
  // switch(cudaStatus) {
  //   case hipSuccess:
  //     std::cout << "[Cuda - Status] " << "Cuda success" << std::endl;
  //     return;
  //   case hipErrorOutOfMemory:
  //     std::cout << "[Cuda - Status] " << "Cuda error memory allocation" << std::endl;
  //     return;
  //   case hipErrorInvalidDevice:
  //   std::cout << "[Cuda - Status] " << "Cuda error invalid device" << std::endl;
  //     return;
  //   case hipErrorInvalidValue:
  //   std::cout << "[Cuda - Status] " << "Cuda error invalid value" << std::endl;
  //     return;
  //   case hipErrorInvalidHandle:
  //   std::cout << "[Cuda - Status] " << "Cuda error invalid resource handle" << std::endl;
  //     return;
  //   case hipErrorUnknown:
  //   std::cout << "[Cuda - Status] " << "Cuda error unknown" << std::endl;
  //     return;
  // }
  // std::cout << "Unregistered error: " << cudaStatus << std::endl;
}

namespace KDriver {
    void defaultKernel (hipGraphicsResource **vbo_res_, dim3 grid, dim3 block, float time) {
      float4 *dptr;
      checkCudaErrors(hipGraphicsMapResources(1, vbo_res_, 0));
      size_t num_bytes;
      hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
        *vbo_res_);
      _default_kernel<<<grid, block>>>(dptr, 256, 256, time);
      hipGraphicsUnmapResources(1, vbo_res_, 0);
    }

    Device::Device() {
      hipGetDevice(&device_);
      hipGetDeviceProperties(&properties_, device_);
      dim3 block(8, 8, 1);
      dim3 grid(256 / block.x, 256 / block.y, 1);
    }

    void Device::setDefaultKernel() {
      kernel_ = defaultKernel;
    }

    void Device::makeGLBuffer(unsigned& buffer) {
      std::cout << "make gl buffer" << std::endl;
      checkCudaErrors(hipGraphicsGLRegisterBuffer(&vbo_res_, buffer, cudaGraphicsMapFlagsWriteDiscard));
      std::cout << "end cuda graphics gl register buffer" << std::endl;
    }

    void Device::setGLDevice() {
      std::cout << "make gl device" << std::endl;
      checkCudaErrors(cudaGLSetGLDevice(device_));
      std::cout << "end cuda gl set gl device" << std::endl;
    }

    void Device::run(float time) {
      kernel_(&vbo_res_, grid_, block_, time);
    }
}
